#include "hip/hip_runtime.h"

// Question answers:
//
// 1 ) int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int index = indx + indy * blockDim.x;

#include <stdio.h>

const int N = 16; 
const int blocksize = 16; 

__global__
void mat_mul(float* a, float* b, float* c, float* input) 
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

int main()
{
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

	float *ad, *bd, *cd;
	const int size = N*N*sizeof(float);

	hipMalloc( (void**)&ad, size );
	hipMalloc( (void**)&bd, size );	
	hipMalloc( (void**)&cd, size );
	
	dim3 dimBlock( blocksize*blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	
	hipMemcpy( ad, a, size, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, size, hipMemcpyHostToDevice ); 
	
	mat_mul<<<dimGrid, dimBlock>>>(ad, bd, cd, N);
	hipDeviceSynchronize();
	
	// cudaMemCpy(dest, src, datasize, arg)
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 

	hipFree( ad );
	hipFree( bd );
	hipFree( cd );

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			printf("%f ", c[i+j*N]);
		}

	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
