#include "hip/hip_runtime.h"

// Question answers:
//
// 1 ) int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int index = indx + indy * blockDim.x;
//
// 2 ) CUDA: N=16 blz=16 0.06
//	     N=32	 0.066
//	     N=64	 0.077
//	     N=128	 0.126
//	     N=256 	 0.276
//	     N=512	 1.341
//	     N=1024	 2.972

// 	CPU: N=16	 0.004
//	     N=32	 0.013
//	     N=64	 0.068
//	     N=128	 0.328
//	     N=256	 2.334

// 3 ) N=16 blz=16 0.06
//    N=16 blz=32 0.05
//    N=16 blz=64 0.04
//    N=16 blz=128 0.04

// 4 ) 0.293 vs 0.276

#include <stdio.h>

const int N = 256; 
const int blocksize = 16; 

__global__
void mat_add(float* a, float* b, float* c) 
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

int main()
{
	
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

	float *ad, *bd, *cd;
	const int size = N*N*sizeof(float);

	hipMalloc( (void**)&ad, size );
	hipMalloc( (void**)&bd, size );	
	hipMalloc( (void**)&cd, size );
	
	dim3 dimBlock( blocksize*blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	
	hipMemcpy( ad, a, size, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, size, hipMemcpyHostToDevice ); 
	
	hipEvent_t e_start;
	hipEventCreate(&e_start);
	hipEventRecord(e_start, 0);

	mat_add<<<dimGrid, dimBlock>>>(ad, bd, cd);
	hipDeviceSynchronize();
	
	// cudaMemCpy(dest, src, datasize, arg)
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 

	hipEvent_t e_stop;
	hipEventCreate(&e_stop);
	hipEventRecord(e_stop, 0);

	hipEventSynchronize(e_start);
	hipEventSynchronize(e_stop);

	hipFree( ad );
	hipFree( bd );
	hipFree( cd );

	float time;
	hipEventElapsedTime(&time, e_start, e_stop);


	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			printf("%f ", c[i+j*N]);
		}

	printf("\n");
	delete[] c;
	
	printf("done, time: %f \n", time);
	return EXIT_SUCCESS;
}
