#include "hip/hip_runtime.h"

// Question answers:
//
// 1 ) int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int index = indx + indy * blockDim.x;

#include <stdio.h>

const int N = 16; 
const int blocksize = 16; 

__global__
void mat_add(float* a, float* b, float* c, float* input) 
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

int main()
{
	hipEvent_t e_start;
	hipEventCreate(&e_start);
	hipEventRecord(e_start, 0);

	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

	float *ad, *bd, *cd;
	const int size = N*N*sizeof(float);

	hipMalloc( (void**)&ad, size );
	hipMalloc( (void**)&bd, size );	
	hipMalloc( (void**)&cd, size );
	
	dim3 dimBlock( blocksize*blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	
	hipMemcpy( ad, a, size, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, size, hipMemcpyHostToDevice ); 
	
	mat_mul<<<dimGrid, dimBlock>>>(ad, bd, cd, N);
	hipDeviceSynchronize();
	
	// cudaMemCpy(dest, src, datasize, arg)
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 

	hipFree( ad );
	hipFree( bd );
	hipFree( cd );

	hipEvent_t e_stop;
	hipEventCreate(&e_stop);
	hipEventRecord(e_stop, 0);

	hipEventSynchronize(e_start);
	hipEventSynchronize(e_stop);

	float time;
	hipEventElapsedTime(&time, e_start, e_stop);


	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			printf("%f ", c[i+j*N]);
		}

	printf("\n");
	delete[] c;
	
	printf("done, time: %f \n", time);
	return EXIT_SUCCESS;
}
